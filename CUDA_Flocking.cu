#include "hip/hip_runtime.h"
#include "CudaFlocking.h"

#include "Graphics.h"
#include "Helper.h"

Graphics graphics;

int main(int argc, char **argv)
{
	startApplication(argc, argv);
	glutMainLoop();
	endApplication();
}

void startApplication(int argc, char **argv)
{
	pArgc = &argc;
	pArgv = argv;
	printf("%s starting...\n", graphics.windowTitle);
	sdkCreateTimer(&timer);
	graphics.initialize(&argc, argv);
	registerGlutCallbacks();
	preparePositionsAndVelocitiesArray();
	prepareObstacles();
	prepareGraphicsToRenderBoids(&vbo);
	prepareCUDADataStructures();
}

void registerGlutCallbacks()
{
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutCloseFunc(cleanup);
}

void preparePositionsAndVelocitiesArray()
{
	for (int i = 0; i < numberOfBoids; i++)
	{
		int a = Helper::randomMinusOneOrOneInt();
		int b = Helper::randomMinusOneOrOneInt();
		velocities[i] = make_float2(a*(float)(rand() % 10) / 50, b*(float)(rand() % 10) / 50);
		velocities[i] = normalizeVector(velocities[i]);
		positions[i] = make_float2(Helper::randomMinusOneOrOneFloat(), Helper::randomMinusOneOrOneFloat());
	}
}

void prepareObstacles()
{
	for (int i = 0; i < numberOfObstacles; i++)
	{
		obstacleCenters[i] = make_float2(Helper::randomMinusOneOrOneFloat() / 2, Helper::randomMinusOneOrOneFloat() / 2);
		obstacleRadii[i] = obstacleRadius;
	}
}

void prepareGraphicsToRenderBoids(GLuint *vbo)
{
	graphics.createGLStructures(vbo, &VAO); 
	graphics.saveBoidsRenderingData(vbo, boidVertices, numberOfBoids); 
	graphics.loadBoidsVertices(vbo); 
	graphics.loadBoidsColor(vbo); 
	graphics.loadBoidsPosition(vbo, &translationsVBO, positions, numberOfBoids); 
	graphics.allowInstancing(); 
}

void prepareCUDADataStructures()
{
	prepareBoidCUDADataStructures();
	prepareObstaclesCUDADataStructures();
}

void prepareBoidCUDADataStructures()
{
	hipMalloc((void**)&dev_positions, numberOfBoids * sizeof(float2));
	hipMemcpy(dev_positions, positions, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_velocities, numberOfBoids * sizeof(float2));
	hipMemcpy(dev_velocities, velocities, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
}

void prepareObstaclesCUDADataStructures()
{
	hipMalloc((void**)&dev_obstacleCenters, numberOfObstacles * sizeof(float2));
	hipMemcpy(dev_obstacleCenters, obstacleCenters, numberOfObstacles * sizeof(float2), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_obstacleRadii, numberOfObstacles * sizeof(float));
	hipMemcpy(dev_obstacleRadii, obstacleRadii, numberOfObstacles * sizeof(float), hipMemcpyHostToDevice);
}

void startOfFrame()
{
	sdkStartTimer(&timer);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glBindVertexArray(VAO);
}

void endOfFrame()
{
	glutSwapBuffers();
	sdkStopTimer(&timer);
	computeFPS();
}

void display()
{
	startOfFrame(); 
	graphics.drawObstacles(numberOfObstacles, obstacleCenters, obstacleRadii);
	calculateBoidsPositions();
	graphics.drawBoids(numberOfBoids,&translationsVBO, positions);
	endOfFrame(); 
}

void calculateBoidsPositions()
{
	updatePositionsWithVelocities << <numberOfBoids / 256 + 1, 256 >> > (dev_positions, dev_velocities, boidRadius, dev_obstacleCenters, dev_obstacleRadii);
	hipMemcpy(positions, dev_positions, numberOfBoids * sizeof(float2), hipMemcpyDeviceToHost);
}

__global__  void updatePositionsWithVelocities(float2 *positions, float2 *velocities, float boidradius, float2 *obstacleCenters, float *obstacleRadii)
{
	unsigned int boidIndex = blockIdx.x*blockDim.x + threadIdx.x;
	if (boidIndex < numberOfBoids)
	{
		float2 alignmentVector = alignment(boidIndex, positions, velocities, boidradius);
		float2 cohesionVector = cohesion(boidIndex, positions, velocities, boidradius);
		float2 separationVector = separation(boidIndex, positions, velocities, boidradius);
		float2 obstacleAvoidanceVector = obstacleAvoidance(positions[boidIndex], velocities[boidIndex], obstacleCenters, obstacleRadii);
		velocities[boidIndex] = calculateBoidVelocity(velocities[boidIndex], alignmentVector,
			cohesionVector, separationVector, obstacleAvoidanceVector);
		positions[boidIndex].x += velocities[boidIndex].x;
		positions[boidIndex].y += velocities[boidIndex].y;
		screenOverflow(positions, boidIndex);
	}
}

__device__ void screenOverflow(float2 *positions, int boidIndex)
{
	float limit = 0.99;
	if (positions[boidIndex].x > limit || positions[boidIndex].x < -limit)
	{
		positions[boidIndex].x *= -1;
	}
	if (positions[boidIndex].y > limit || positions[boidIndex].y < -limit)
	{
		positions[boidIndex].y *= -1;
	}
}

void endApplication()
{
	freeCUDADataStructures();
	printf("%s completed, returned %s\n", graphics.windowTitle, (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void freeCUDADataStructures()
{
	hipFree(dev_positions);
	hipFree(dev_velocities);
	hipFree(dev_obstacleCenters);
	hipFree(dev_obstacleRadii);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;
	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}
	char fps[256];
	sprintf(fps, "CUDA Flock: %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{
	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
}

void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
	}
}

void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
		sendFlockToMouseClick(x, y);
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void sendFlockToMouseClick(int x, int y)
{
	float2 destination = mouseToWorldCoordinates(x, y);
	setFlockDestination(destination);
}

float2 mouseToWorldCoordinates(int x, int y)
{
	float fX = (float)x / window_width;
	float fY = (float)y / window_width;
	fX = fX * 2 - 1;
	fY = -fY * 2 + 1;
	return make_float2(fX, fY);
}

void setFlockDestination(float2 destination)
{
	for (int i = 0; i < numberOfBoids; i++)
	{
		velocities[i].x += destination.x - positions[i].x;
		velocities[i].y += destination.y - positions[i].y;
		velocities[i] = normalizeVector(velocities[i]);
	}
	hipMemcpy(dev_velocities, velocities, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
}
