#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
	This example demonstrates how to use the Cuda OpenGL bindings to
	dynamically modify a vertex buffer using a Cuda kernel.

	The steps are:
	1. Create an empty vertex buffer object (VBO)
	2. Register the VBO with Cuda
	3. Map the VBO for writing from Cuda
	4. Run Cuda kernel to modify the vertex positions
	5. Unmap the VBO
	6. Render the results using OpenGL

	Host code
*/


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>
#include <shader.h>


// includes, cuda
#include <hip/hip_runtime.h>
#include ""
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

GLuint instanceVBO;

//vao variables
unsigned int VAO;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

float2 translations[100];

Shader* shPointer;

int movementTime = 1; 
int timecount = 0; 


#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *windowTitle = "CUDA_Flocking (VBO)";


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

	pArgc = &argc;
	pArgv = argv;

#if defined(__linux__)
	setenv("DISPLAY", ":0", 0);
#endif

	printf("%s starting...\n", windowTitle);

	runTest(argc, argv);

	printf("%s completed, returned %s\n", windowTitle, (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "CUDA Flock: %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();

	SDK_CHECK_ERROR_GL();

	Shader shader("shaders/flock.vert", "shaders/flock.frag");
	shader.use(); 
	shPointer = (Shader*)malloc(sizeof(Shader)); 
	shPointer = &shader;

	return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv)
{
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}

	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutCloseFunc(cleanup);

	// create VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard); // il flag indica che CUDA non legger� dalla risorsa

	// run the cuda part
	runCuda(&cuda_vbo_resource);

	// start rendering mainloop
	glutMainLoop();


	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
	// map OpenGL buffer object for writing from CUDA
	float4 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		*vbo_resource));

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}



void massMovement(bool random = false)
{
	int index = 0;
	float offset = 0.001f;

	for (int y = -10; y < 10; y += 2)
	{
		for (int x = -10; x < 10; x += 2)
		{
			float2 translation;
			if (!random) {
				translation.x = (float)x / 10.0f + offset;
				translation.y = (float)y / 10.0f + offset;
			}

			else {
				translation.x = (rand() % 2 * 2 - 1)  * offset;
				translation.y = (rand() % 2 * 2 - 1)  * offset;
			}
			index++; 
			translations[index].x += translation.x;
			translations[index].y += translation.y;
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags)
{

	float quadVertices[] = {
		// positions     // colors
		-0.05f,  0.05f,  1.0f, 0.0f, 0.0f,
		0.05f, -0.05f,  0.0f, 1.0f, 0.0f,
		-0.05f, -0.05f,  0.0f, 0.0f, 1.0f
	};
	
	massMovement(true); 

	assert(vbo);

	//vertices vbo
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, vbo);
	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, *vbo);
	glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), &quadVertices[0], GL_DYNAMIC_DRAW);

	//loading positions of vertices
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	//loading colors
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)3); 
	glEnableVertexAttribArray(1); 

	//loading offsets
	glGenBuffers(1, &instanceVBO);
	glBindBuffer(GL_ARRAY_BUFFER, instanceVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float2) * 100, &translations[0], GL_DYNAMIC_DRAW);

	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 0, (void*)0);
	glEnableVertexAttribArray(2);

	glVertexAttribDivisorARB(2, 1);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

	// unregister this buffer object with CUDA
	checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback -> called multiple times after the first glutMainLoop() 
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	//runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glBindVertexArray(VAO);

	timecount++; 
	if (timecount >= movementTime) {
		massMovement(true); 
		glBindBuffer(GL_ARRAY_BUFFER, instanceVBO);
		glBufferData(GL_ARRAY_BUFFER, sizeof(float2) * 100, &translations[0], GL_DYNAMIC_DRAW);
		timecount = 0;
	}

	glDrawArraysInstanced(GL_TRIANGLES, 0, 6, 100);

	glutSwapBuffers();

	sdkStopTimer(&timer);
	computeFPS();
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
	if (!d_vbo_buffer)
	{
		checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

		// map buffer object
		glBindBuffer(GL_ARRAY_BUFFER, vbo);
		float *data = (float *)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

		// check result
		if (checkCmdLineFlag(argc, (const char **)argv, "regression"))
		{
			// write file for regression test
			sdkWriteFile<float>("./data/regression.dat",
				data, mesh_width * mesh_height * 3, 0.0, false);
		}

		// unmap GL buffer object
		if (!glUnmapBuffer(GL_ARRAY_BUFFER))
		{
			fprintf(stderr, "Unmap buffer failed.\n");
			fflush(stderr);
		}

		checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
			cudaGraphicsMapFlagsWriteDiscard));

		SDK_CHECK_ERROR_GL();
	}
}
