#include "hip/hip_runtime.h"
#include "Utilities.h"

int main(int argc, char **argv)
{
	startApplication(argc, argv);
	glutMainLoop();
	endApplication();
}

void startApplication(int argc, char **argv)
{
	pArgc = &argc;
	pArgv = argv;
	printf("%s starting...\n", windowTitle);
	sdkCreateTimer(&timer);
	initGL(&argc, argv);
	registerGlutCallbacks();
	preparePositionsAndVelocitiesArray();
	createVBO(&vbo);
	prepareCUDADataStructures();
}

bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();

	SDK_CHECK_ERROR_GL();

	Shader shader("shaders/flock.vert", "shaders/flock.frag");
	shader.use();
	shPointer = (Shader*)malloc(sizeof(Shader));
	shPointer = &shader;

	return true;
}

void registerGlutCallbacks()
{
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutCloseFunc(cleanup);
}

void preparePositionsAndVelocitiesArray()
{
	for (int i = 0; i < numberOfBoids; i++)
	{
		int a = randomMinusOneOrOneInt();
		int b = randomMinusOneOrOneInt();
		velocities[i] = make_float2(a*(float)(rand() % 10) / 50, b*(float)(rand() % 10) / 50);
		velocities[i] = normalizeVector(velocities[i]);
		positions[i] = make_float2(randomMinusOneOrOneFloat(), randomMinusOneOrOneFloat());
	}
}

void createVBO(GLuint *vbo)
{
	assert(vbo);

	//vertices vbo
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, vbo);
	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, *vbo);
	glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), &quadVertices[0], GL_DYNAMIC_DRAW);

	//loading positions of vertices
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	//loading colors
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)3);
	glEnableVertexAttribArray(1);

	//loading position offsets
	glGenBuffers(1, &translationsVBO);
	glBindBuffer(GL_ARRAY_BUFFER, translationsVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float2) * numberOfBoids, &positions[0], GL_DYNAMIC_DRAW);

	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 0, (void*)0);
	glEnableVertexAttribArray(2);

	//this is necessary for instancing in openGL
	glVertexAttribDivisorARB(2, 1);

	SDK_CHECK_ERROR_GL();
}

void prepareCUDADataStructures()
{
	hipMalloc((void**)&dev_positions, numberOfBoids * sizeof(float2));
	hipMemcpy(dev_positions, positions, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_velocities, numberOfBoids * sizeof(float2));
	hipMemcpy(dev_velocities, velocities, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
}

void display()
{
	sdkStartTimer(&timer);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glBindVertexArray(VAO);
	timecount++;
	if (timecount >= movementTime) {
		launchFlockingKernel();
		timecount = 0;
	}
	loadPositionOffsetOnVBO();
	glDrawArraysInstanced(GL_TRIANGLES, 0, 3, numberOfBoids);
	glutSwapBuffers();
	sdkStopTimer(&timer);
	computeFPS();
}

void launchFlockingKernel()
{
	updatePositionsWithVelocities << <1, numberOfBoids >> > (dev_positions, dev_velocities, boidRadius);
	hipMemcpy(positions, dev_positions, numberOfBoids * sizeof(float2), hipMemcpyDeviceToHost);
}

__global__  void updatePositionsWithVelocities(float2 *positions, float2 *velocities, float boidradius)
{
	unsigned int boidIndex = blockIdx.x*blockDim.x + threadIdx.x;
	if (boidIndex < numberOfBoids)
	{
		float2 alignmentVector = alignment(boidIndex, positions, velocities, boidradius);
		float2 cohesionVector = cohesion(boidIndex, positions, velocities, boidradius);
		float2 separationVector = separation(boidIndex, positions, velocities, boidradius);
		velocities[boidIndex] = calculateBoidVelocity(velocities[boidIndex], alignmentVector,
			cohesionVector, separationVector);
		positions[boidIndex].x += velocities[boidIndex].x;
		positions[boidIndex].y += velocities[boidIndex].y;
	}
}

void loadPositionOffsetOnVBO()
{
	glBindBuffer(GL_ARRAY_BUFFER, translationsVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float2) * numberOfBoids, &positions[0], GL_DYNAMIC_DRAW);
}

void endApplication()
{
	freeCUDADataStructures();
	printf("%s completed, returned %s\n", windowTitle, (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void freeCUDADataStructures()
{
	hipFree(dev_positions);
	hipFree(dev_velocities);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;
	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}
	char fps[256];
	sprintf(fps, "CUDA Flock: %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}

int randomMinusOneOrOneInt()
{
	return (int)rand() % 2 * 2 - 1;;
}

float randomMinusOneOrOneFloat()
{
	return (float)(rand() % 101) / 100 * 2 - 1;;
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{
	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
}

void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
}
}

void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
		sendFlockToMouseClick(x, y);
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void sendFlockToMouseClick(int x, int y)
{
	float2 destination = mouseToWorldCoordinates(x, y);
	setFlockDestination(destination);
}

float2 mouseToWorldCoordinates(int x, int y)
{
	float fX = (float)x / window_width;
	float fY = (float)y / window_width;
	fX = fX * 2 - 1;
	fY = -fY * 2 + 1;
	return make_float2(fX, fY);
}

void setFlockDestination(float2 destination)
{
	for (int i = 0; i < numberOfBoids; i++)
	{
		velocities[i].x = destination.x - positions[i].x;
		velocities[i].y = destination.y - positions[i].y;
	}
	hipMemcpy(dev_velocities, velocities, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
}
