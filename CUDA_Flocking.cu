#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
	This example demonstrates how to use the Cuda OpenGL bindings to
	dynamically modify a vertex buffer using a Cuda kernel.

	The steps are:
	1. Create an empty vertex buffer object (VBO)
	2. Register the VBO with Cuda
	3. Map the VBO for writing from Cuda
	4. Run Cuda kernel to modify the vertex positions
	5. Unmap the VBO
	6. Render the results using OpenGL

	Host code
*/


#include "Boid.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>
#include <shader.h>


// includes, cuda
#include <hip/hip_runtime.h>
#include ""
#include <cuda_gl_interop.h>
#include "hip/device_functions.h"; 

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#include "DeviceFunctions.h"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

GLuint translationsVBO;

//vao variables
unsigned int VAO;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

Shader* shPointer;

float2 *pos;

int movementTime = 1;
int timecount = 0;

//boid i is defined by positions[i] and velocities[i]
float2 positions[numberOfBoids];
float2 velocities[numberOfBoids];

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

float2 *dev_positions, *dev_velocities;

const char *windowTitle = "CUDA_Flocking";

__device__ float2 alignment(int threadX, float2 *positions, float2 *velocities, float boidradius)
{
	float2 alignmentVector = make_float2(0, 0);
	int cont = 0;
	for (int i = 0; i < numberOfBoids; i++)
	{
		float2 point1, point2;
		point1 = positions[threadX];
		point2 = positions[i];

		if (threadX != i &&  distanceBetweenPoints(point1, point2) < boidradius)
		{
			alignmentVector.x += velocities[i].x;
			alignmentVector.y += velocities[i].y;
			cont++;
		}
	}

	alignmentVector = vectorDivision(alignmentVector, cont);

	alignmentVector = normalizeVector(alignmentVector); 

	return alignmentVector;
}

__device__ float2 cohesion(int threadX, float2 *positions, float2 *velocities, float boidradius)
{
	float2 cohesionVector = make_float2(0, 0);
	int cont = 0;

	for (int i = 0; i < numberOfBoids; i++)
	{
		float2 point1, point2;
		point1 = positions[threadX];
		point2 = positions[i];
		float distance = sqrtf(pow(point2.x - point1.x, 2) + pow(point2.y - point1.y, 2));

		if (threadX != i &&  distance < boidradius)
		{
			cohesionVector.x += positions[i].x;
			cohesionVector.y += positions[i].y;
			cont++;
		}
	}

	cohesionVector = vectorDivision(cohesionVector, cont); 

	cohesionVector.x -= positions[threadX].x;
	cohesionVector.y -= positions[threadX].y;

	cohesionVector = normalizeVector(cohesionVector); 

	return cohesionVector;
}

__device__ float2 separation(int threadX, float2 *positions, float2 *velocities, float boidradius)
{
	float2 separationVector = make_float2(0, 0);
	int cont = 0;

	for (int i = 0; i < numberOfBoids; i++)
	{
		float2 point1, point2;
		point1 = positions[threadX];
		point2 = positions[i];
		float distance = distanceBetweenPoints(point1, point2); 

		if (threadX != i &&  distance < boidradius)
		{
			separationVector.x += positions[i].x - positions[threadX].x;
			separationVector.y += positions[i].y - positions[threadX].y;
			cont++;
		}

	}

	separationVector.x /= cont;
	separationVector.y /= cont;

	separationVector.x *= -1;
	separationVector.y *= -1;	

	///normalization of separation
	separationVector = normalizeVector(separationVector);

	return separationVector;
}

__global__  void updatePositionsWithVelocities(float2 *positions, float2 *velocities, float boidradius)
{
	unsigned int threadX = blockIdx.x*blockDim.x + threadIdx.x;

	float alignmentWeight, cohesionWeight, separationWeight; 

	alignmentWeight = 9; 
	cohesionWeight = 4;
	separationWeight = 4; 

	float boidSpeed = 0.005;

	if (threadX < numberOfBoids)
	{
		float2 alignmentVector = alignment(threadX, positions, velocities, boidradius) ;
		float2 cohesionVector = cohesion(threadX, positions, velocities, boidradius);
		float2 separationVector = separation(threadX, positions, velocities, boidradius);

		float2 velocityOfTheBoid = velocities[threadX];
		velocityOfTheBoid.x += alignmentVector.x * alignmentWeight 
			+ cohesionVector.x * cohesionWeight 
			+ separationVector.x * separationWeight;

		velocityOfTheBoid.y += alignmentVector.y * alignmentWeight 
			+ cohesionVector.y * cohesionWeight
			+ separationVector.y * separationWeight;

		velocityOfTheBoid = normalizeVector(velocityOfTheBoid);
		velocityOfTheBoid = vectorMultiplication(velocityOfTheBoid, boidSpeed); 
		velocities[threadX] = velocityOfTheBoid;

		positions[threadX].x += velocities[threadX].x;
		positions[threadX].y += velocities[threadX].y;
	}
}

void launchKernel()
{
	updatePositionsWithVelocities << <1, 512 >> > (dev_positions, dev_velocities, boidRadius);
	hipMemcpy(positions, dev_positions, numberOfBoids * sizeof(float2), hipMemcpyDeviceToHost);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

	pArgc = &argc;
	pArgv = argv;

	printf("%s starting...\n", windowTitle);

	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	initGL(&argc, argv);

	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutCloseFunc(cleanup);


	for (int i = 0; i < numberOfBoids; i++)
	{
		int a = rand() % 2 * 2 - 1;
		int b = rand() % 2 * 2 - 1;
		
		velocities[i] = make_float2(a*(float)(rand() % 10) / 50, b*(float)(rand() % 10) / 50);
		velocities[i] = normalizeVector(velocities[i]); 
		//velocities[i] = make_float2(0.01, 0.01);
		//positions[i] = make_float2((float) 0.0f, (float) 0.0f); 
		positions[i] = make_float2(a*(float)(rand() % 10) / 50, b*(float)(rand() % 10) / 50);
	}

	createVBO(&vbo);

	hipMalloc((void**)&dev_positions, numberOfBoids * sizeof(float2));
	hipMalloc((void**)&dev_velocities, numberOfBoids * sizeof(float2));

	hipMemcpy(dev_positions, positions, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(dev_velocities, velocities, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);

	// start rendering mainloop
	glutMainLoop();

	hipFree(dev_positions);
	hipFree(dev_velocities);

	printf("%s completed, returned %s\n", windowTitle, (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "CUDA Flock: %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();

	SDK_CHECK_ERROR_GL();

	Shader shader("shaders/flock.vert", "shaders/flock.frag");
	shader.use();
	shPointer = (Shader*)malloc(sizeof(Shader));
	shPointer = &shader;

	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO and fills the VBO so that the positions of the boids can be modifiable
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo)
{
	float quadVertices[] = {
		// positions     // colors
		0.0f,  0.05f,  1.0f, 0.0f, 0.0f,
		0.02f, -0.02f,  0.0f, 1.0f, 0.0f,
		-0.02f, -0.02f,  0.0f, 0.0f, 1.0f
	};

	assert(vbo);

	//vertices vbo
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, vbo);
	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, *vbo);
	glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), &quadVertices[0], GL_DYNAMIC_DRAW);

	//loading positions of vertices
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	//loading colors
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)3);
	glEnableVertexAttribArray(1);

	//loading position offsets
	glGenBuffers(1, &translationsVBO);
	glBindBuffer(GL_ARRAY_BUFFER, translationsVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float2) * numberOfBoids, &positions[0], GL_DYNAMIC_DRAW);

	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 0, (void*)0);
	glEnableVertexAttribArray(2);

	//this is necessary for instancing in openGL
	glVertexAttribDivisorARB(2, 1);

	SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{
	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Display callback -> called multiple times after the first glutMainLoop() 
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glBindVertexArray(VAO);

	timecount++;

	if (timecount >= movementTime) {
		launchKernel();
		timecount = 0;
	}

	////loading position offsets
	glBindBuffer(GL_ARRAY_BUFFER, translationsVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float2) * numberOfBoids, &positions[0], GL_DYNAMIC_DRAW);

	glDrawArraysInstanced(GL_TRIANGLES, 0, 3, numberOfBoids);

	glutSwapBuffers();

	sdkStopTimer(&timer);
	computeFPS();
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
	}
}

float2 mouseToWorldCoordinates(int x, int y)
{
	float fX = (float)x / window_width;
	float fY = (float)y / window_width; 
	fX = fX * 2 - 1; 
	fY = -fY * 2 +1; 
	return make_float2(fX, fY); 
}

void setFlockDestination(float2 destination)
{
	for (int i = 0; i < numberOfBoids; i++)
	{
		velocities[i].x = destination.x - positions[i].x; 
		velocities[i].y = destination.y - positions[i].y;
	}
	hipMemcpy(dev_velocities, velocities, numberOfBoids * sizeof(float2), hipMemcpyHostToDevice);
}


////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
		float2 destination = mouseToWorldCoordinates(x, y); 
		setFlockDestination(destination); 
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}


void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}